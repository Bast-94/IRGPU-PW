
#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char **argv) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  std::cerr << "Getting GPU Data." << std::endl;
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        std::cerr << "No CUDA GPU has been detected" << std::endl;
        return -1;
      } else if (deviceCount == 1) {
        std::cerr << "There is 1 device supporting CUDA" << std::endl;
      } else {
        std::cerr << "There are " << deviceCount << " devices supporting CUDA"
                  << std::endl;
      }
    }
    std::cerr << "Device " << dev 
              << " name: " << deviceProp.name
              << std::endl;
    std::cerr << " Computational Capabilities: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << "."
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum global memory size: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum constant memory size: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum shared memory size per block: "
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum block dimensions: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << " x " << "FIXME" //@@ Print appropriate `deviceProp` field
              << " x " << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum grid dimensions: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << " x " << "FIXME" //@@ Print appropriate `deviceProp` field
              << " x " << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Warp size: " 
              << "FIXME" //@@ Print appropriate `deviceProp` field
              << std::endl;
  }
  std::cerr << "End of GPU data gathering." << std::endl;
  return 0;
}
